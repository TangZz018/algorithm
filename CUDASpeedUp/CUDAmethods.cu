#include "hip/hip_runtime.h"
//
// Created by HP on 2023/12/31.
//
#include <cstdio>
#include <cmath>
#include "CUDAmethods.cuh"
#include <hip/hip_runtime.h>
#include "immintrin.h"
#include <windows.h>
extern float *hostInput, *hostOutput;      // host
extern float *deviceInput, *deviceOutput;  // GPU
extern LARGE_INTEGER start, end;
const int data_len = DATANUM;
////////////////////////////////
//            CPU             //
////////////////////////////////
float sum(const float data[], const int len)
{
    float sum = 0.0;
    float c = 0.0;

    for (int i = 0; i < len; i++) {
        float y = log(sqrt(data[i])) - c;
        float t = sum + y;
        c = (t - sum) - y;
        sum = t;

    }
    return sum;
}

float floatMax(const float data[], const int len)
{
    float max = 1;
    int max_id = 0;
    float sqrt_max = sqrt(max);
    float log_sqrt_max = log(sqrt_max);
    for (int i = 0; i < len; i++)
    {
        float sqrt_data = sqrt(data[i]);
        float log_sqrt_data = log(sqrt_data);

        if (log_sqrt_data > log_sqrt_max)
        {
            max_id = i;
            max = data[i];
            sqrt_max = sqrt_data;
            log_sqrt_max = log_sqrt_data;
        }
    }
    return log(sqrt(max));
}

void merge(float *list, float *sorted, int start, int mid, int end)
{
    int ti=start, i=start, j=mid;
    while (i<mid || j<end)
    {
        if (j==end)
            sorted[ti] = list[i++];
        else if (i==mid)
            sorted[ti] = list[j++];
        else if (list[i]<list[j])
            sorted[ti] = list[i++];
        else
            sorted[ti] = list[j++];
        ti++;
    }

    for (ti=start; ti<end; ti++)
        list[ti] = sorted[ti];
}

void mergesort_recur(float *list, float *sorted, int start, int end)
{
    if (end-start<2)
        return;
    mergesort_recur(list, sorted, start, start + (end-start)/2);
    mergesort_recur(list, sorted, start + (end-start)/2, end);
    merge(list, sorted, start, start + (end-start)/2, end);
}

int mergesort_cpu(const float *list, float *sorted, int n)
{
    float* arr = new float[DATANUM];
    for(int i=0;i<n;i++)
        arr[i]= log(sqrt(list[i]));

    mergesort_recur(arr, sorted, 0, n);
    return 1;
}

////////////////////////////////
//            GPU             //
////////////////////////////////

void syncAndCheckCUDAError() {
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(cudaStatus));
    }
}

__global__ void sum1(const float *src, int size, float *result){
    __shared__ float cache[THREADS_PER_BLOCK];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cachedIndex = threadIdx.x;
    float temp = 0;
    while(tid < size) {
        temp += log(sqrt(src[tid]));
        tid += gridDim.x * blockDim.x;
    }
    cache[cachedIndex] = temp;

    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cachedIndex < i) {
            cache[cachedIndex] += cache[cachedIndex + i];
        }
        __syncthreads();
    }

    if(cachedIndex == 0)
        atomicAdd(result, cache[0]);
}

__global__ void sum2(float *result) {
    int cachedIndex = threadIdx.x;

    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cachedIndex < i) {
            result[cachedIndex] += result[cachedIndex + i];
        }
        __syncthreads();
    }
}

float sumSpeedUpCUDA(){
    int block_num = (DATANUM + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    if (block_num > THREADS_PER_BLOCK)
        block_num = THREADS_PER_BLOCK;
    QueryPerformanceCounter(&start);
    sum1<<<block_num, THREADS_PER_BLOCK>>>(deviceInput, DATANUM, deviceOutput);
    sum2<<<1, block_num>>>(deviceOutput);
    hipDeviceSynchronize();
    QueryPerformanceCounter(&start);
    float result = 0;
    hipMemcpy(&result, deviceOutput, sizeof(float), hipMemcpyDeviceToHost);
    syncAndCheckCUDAError();

    hipFree(deviceInput);
    hipFree(deviceOutput);
    syncAndCheckCUDAError();

    return result;
}

__device__ void atomicMaxFloat(float* address, float val) {
    int* address_as_int = reinterpret_cast<int*>(address);
    int old_val_as_int = *address_as_int;
    int assumed;
    do {
        assumed = old_val_as_int;
        float old_val = __int_as_float(assumed);
        float new_val = fmaxf(old_val, val);
        old_val_as_int = __float_as_int(new_val);
    } while (assumed != old_val_as_int);
    *address_as_int = old_val_as_int;
}

__global__ void max_gpu(const float* src, int size, float* result) {
    __shared__ float cache[THREADS_PER_BLOCK];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cachedIndex = threadIdx.x;

    float temp = 0;
    while (tid < size) {
        temp = fmaxf(temp, logf(sqrtf(src[tid])));
        tid += gridDim.x * blockDim.x;
    }
    cache[cachedIndex] = temp;

    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cachedIndex < i) {
            cache[cachedIndex] = fmaxf(cache[cachedIndex], cache[cachedIndex + i]);
        }
        __syncthreads();
    }

    if (cachedIndex == 0) {
        atomicMaxFloat(result, cache[0]);
    }
}

float maxSpeedUpCUDA() {
    int block_num = (data_len + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    if (block_num > THREADS_PER_BLOCK)
        block_num = THREADS_PER_BLOCK;

    QueryPerformanceCounter(&start);
    max_gpu<<<block_num, THREADS_PER_BLOCK>>>(deviceInput, data_len, deviceOutput);
    QueryPerformanceCounter(&end);

    float result = 0;
    hipMemcpy(&result, deviceOutput, sizeof(float), hipMemcpyDeviceToHost);
    syncAndCheckCUDAError();

    hipFree(deviceInput);
    hipFree(deviceOutput);
    syncAndCheckCUDAError();

    return result;
}

__device__ void merge_gpu(float *list, float *sorted, int start, int mid, int end)
{
    int k=start, i=start, j=mid;
    while (i<mid || j<end)
    {
        if (j==end) sorted[k] = list[i++];
        else if (i==mid) sorted[k] = list[j++];
        else if (list[i]<list[j]) sorted[k] = list[i++];
        else sorted[k] = list[j++];
        k++;
    }
}

__global__ void mergesort_gpu(float *list, float *sorted, int n, int chunk){

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int start = tid * chunk;
    if(start >= n)
        return;

    int mid = min(start + chunk / 2, n);
    int end = min(start + chunk, n);
    merge_gpu(list, sorted, start, mid, end);
}

// Sequential Merge Sort for GPU when Number of Threads Required gets below 1 Warp Size
void mergesort_gpu_seq( float *list, float *sorted, int n, int chunk){
    int chunk_id;
    for(chunk_id=0; chunk_id*chunk<=n; chunk_id++){
        int start = chunk_id * chunk, end, mid;
        if(start >= n)
            return;
        mid = min(start + chunk/2, n);
        end = min(start + chunk, n);
        merge(list, sorted, start, mid, end);
    }
}

int mergesortCUDA(const float list[], float sorted[], int n){
    float* arr = new float[DATANUM];
    float *list_d;
    float *sorted_d;
    int dummy;
    bool flag = false;
    bool sequential = false;

    int size = n * sizeof(int);

    hipMalloc((void **)&list_d, size);
    hipMalloc((void **)&sorted_d, size);

    hipMemcpy(list_d, arr, size, hipMemcpyHostToDevice);
    hipError_t err = hipGetLastError();
    if(err!=hipSuccess){
        printf("Error_2: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    const int max_active_blocks_per_sm = 32;
    const int max_active_warps_per_sm = 64;

    int warp_size = prop.warpSize;
    int max_grid_size = prop.maxGridSize[0];
    int max_threads_per_block = prop.maxThreadsPerBlock;
    int max_procs_count = prop.multiProcessorCount;

    int max_active_blocks = max_active_blocks_per_sm * max_procs_count;
    int max_active_warps = max_active_warps_per_sm * max_procs_count;

    int chunk_size;

    // Time Start
    QueryPerformanceCounter(reinterpret_cast<LARGE_INTEGER *>(&start));
    for(int i=0;i<n;i++)
        arr[i]= log(sqrt(list[i]));
    for(chunk_size=2; chunk_size<2*n; chunk_size*=2){
        int blocks_required=0, threads_per_block=0;
        int threads_required = (n%chunk_size==0) ? n/chunk_size : n/chunk_size+1;

        if (threads_required<=warp_size*3 && !sequential){
            sequential = true;
            //Time End
            QueryPerformanceCounter(reinterpret_cast<LARGE_INTEGER *>(&end));
            if(flag)
                hipMemcpy(sorted, sorted_d, size, hipMemcpyDeviceToHost);
            else
                hipMemcpy(sorted, list_d, size, hipMemcpyDeviceToHost);
            err = hipGetLastError();
            if(err!=hipSuccess){
                printf("ERROR_4: %s\n", hipGetErrorString(err));
                return -1;
            }
            hipFree(list_d);
            hipFree(sorted_d);
        }
        else if (threads_required<max_threads_per_block){
            threads_per_block = warp_size*4;
            dummy = threads_required/threads_per_block;
            blocks_required = (threads_required%threads_per_block==0) ? dummy : dummy+1;
        }
        else if(threads_required<max_active_blocks*warp_size*4){
            threads_per_block = max_threads_per_block/2;
            dummy = threads_required/threads_per_block;
            blocks_required = (threads_required%threads_per_block==0) ? dummy : dummy+1;
        }
        else{
            dummy = threads_required/max_active_blocks;
            int estimated_threads_per_block = (threads_required%max_active_blocks==0) ? dummy : dummy+1;
            if(estimated_threads_per_block > max_threads_per_block){
                threads_per_block = max_threads_per_block;
                dummy = threads_required/max_threads_per_block;
                blocks_required = (threads_required%max_threads_per_block==0) ? dummy : dummy+1;
            }
            else{
                threads_per_block = estimated_threads_per_block;
                blocks_required = max_active_blocks;
            }
        }

        if(blocks_required>=max_grid_size){
            printf("ERROR_2: Too many Blocks Required\n");
            return -1;
        }
        if(sequential){
            mergesort_gpu_seq(arr, sorted, n, chunk_size);
        }else{
            if(flag) mergesort_gpu<<<blocks_required, threads_per_block>>>(sorted_d, list_d, n, chunk_size);
            else mergesort_gpu<<<blocks_required, threads_per_block>>>(list_d, sorted_d, n, chunk_size);
            hipDeviceSynchronize();
            err = hipGetLastError();
            if(err!=hipSuccess){
                printf("ERROR_3: %s\n", hipGetErrorString(err));
                return -1;
            }
            flag = !flag;
        }
    }
    return 0;
}